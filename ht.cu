#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <inttypes.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "opt.h"

#ifdef __GNUC__
#	define PURE __attribute__((pure))
#	define CONST __attribute__((const))
#else
#	define PURE /* no pure function support */
#	define CONST /* no const function support */
#endif

PURE __device__ static uint32_t
serialize(const uint32_t bidx[4], const unsigned bdims[4])
{
	return 1 + bidx[0] + bidx[1]*bdims[0] + bidx[2]*bdims[0]*bdims[1] +
	       bidx[3]*bdims[0]*bdims[1]*bdims[2];
}

__device__ static unsigned next = 0x12345678u;
__device__ static unsigned
devrand()
{
	next = next * 1103515245 + 12345;
	return ((unsigned)(next/2147483648) % 1073741824);
}

/* 16384^3 volume / 32^3 voxel bricks == 512^3 bricks.  So an
 * axis-aligned ray (i.e. a thread) couldn't request more than 512
 * bricks. */
const size_t MAX_BRICK_REQUESTS = 512;

__constant__ unsigned brickdims[4] = {0};

/* try to find the given value in the table.  this may not occur at the hashed
 * position, of course, since collisions may occur.  it would be at subsequent
 * elements, then. */
__device__ static bool
find_entry(unsigned* ht, const size_t htlen, unsigned value)
{
#	define ELEMS_TO_SEARCH 4
	for(size_t i=0; i < ELEMS_TO_SEARCH; ++i) {
		const unsigned idx = (value + i) % htlen;
		if(ht[idx] == value) { return true; }
	}
	return false;
}

/* flushes all the entries from 'pending' to the hash table. */
__device__ static void
flush(unsigned* ht, const size_t htlen, unsigned pending[16], const size_t n)
{
	for(size_t i=0; i < n; ++i) {
		size_t iter = 0;
		do {
			const unsigned hpos = (pending[i] + iter) % htlen;
			uint32_t value = atomicCAS(&ht[hpos], 0U, pending[i]);
			if(value == 0 || value == pending[i]) { break; }
		} while(++iter < 10);
		/* We could atomicExch pending[i] back to 0 now.. but there's
		 * not really a point. */
		/* atomicExch(&pending[i], 0U); */
	}
}


/** @param ht the hash table
 * @param ??? the dimensions of the hash table, in shared mem
 * @param list of bricks to access.  this is 4-components! (x,y,z, LOD) */
__global__ void
ht_inserts(unsigned* ht, const size_t htlen, const uint32_t* bricks,
           const size_t nbricks)
{
	/* shared memory for writes which should get added to 'ht'. */
	__shared__ unsigned pending[16];
	__shared__ unsigned pidx;

	/* __shared__ vars can't have initializers; do it manually. */
	for(size_t i=0; i < 16; i++) { pending[i] = 0; }
	pidx = 0;
	__syncthreads();

	for(size_t i=0; i < MAX_BRICK_REQUESTS; ++i) {
		const unsigned bid = devrand() % nbricks;
		unsigned serialized = serialize(&bricks[bid*4], brickdims);

		/* Is it already in the table?  then move on. */
		if(find_entry(ht, htlen, serialized)) { continue; }

		/* Otherwise, add it to our list of pending writes into the
		 * table.  But, that might cause it to overflow, which means
		 * we'd have to flush it. */
		if(pidx >= 16) {
			flush(ht, htlen, pending, 16);
			atomicCAS(&pidx, 16U, 0U);
		} else {
			atomicExch(&pending[pidx], serialized);
			atomicAdd(&pidx, 1);
		}
	}
	flush(ht, htlen, pending, pidx);
}

__global__ void
ht_inserts_simple(unsigned* ht, const size_t htlen, const uint32_t* bricks,
                  const size_t nbricks)
{
	for(size_t i=0; i < MAX_BRICK_REQUESTS; ++i) {
		const unsigned bid = devrand() % nbricks;
		unsigned serialized = serialize(&bricks[bid*4], brickdims);

		unsigned rehash_count = 0;
		do {
			const unsigned hpos = (serialized + rehash_count) %
			                       htlen;
			unsigned val = atomicCAS(&ht[hpos], 0U, serialized);
			if(val == 0 || val == serialized) { break; }
		} while(++rehash_count < 10);
	}
}

/** reads requests from the given filename.
 * @returns the array of requests, or NULL on error.
 * @param[out] nreqs the number of requests in the array; note the array
 *             then has 4*requests elements, since each requests is 4
 *             entries. */
static uint32_t*
requests_from(const char* filename, size_t* nreqs)
{
	*nreqs = 0;
	FILE* fp = fopen(filename, "r");
	if(NULL == fp) {
		errno = EINVAL;
		return NULL;
	}
	if(fscanf(fp, "%zu\n", nreqs) != 1) {
		fclose(fp);
		errno = EPROTO;
		return NULL;
	}
	uint32_t* requests = (uint32_t*)malloc(sizeof(uint32_t)*4*(*nreqs));
	for(size_t req=0; req < *nreqs; ++req) {
		int scan = fscanf(fp, "%u %u %u %u\n", &requests[req*4+0],
		                  &requests[req*4+1], &requests[req*4+2],
		                  &requests[req*4+3]);
		if(scan != 4) {
			fprintf(stderr, "Error scanning request %zu: %d\n",
			        req, errno);
			fclose(fp);
			free(requests);
			*nreqs = 0;
			errno = EPROTO;
			return NULL;
		}
	}
	return requests;
}

/* are the given requests valid?  they need to fall within brick indices.
 * @param requests the requests the examine
 * @param nreq number of requests; 'requests' is 4*nreq elems long.
 * @param bdims the brick dimensions.
 * @param[out] erridx if nonnull, the request which was in error. */
static bool
requests_verify(const uint32_t* requests, const size_t nreq,
                const unsigned bdims[4], size_t* erridx)
{
	/* this actually isn't great, because we assume that the valid indices
	 * are 0 to bdims[0]*bdims[1]*bdims[2]*bdims[3].  In reality, the
	 * number of bricks decreases by half every time we drop to a coarser
	 * LOD, so there are far fewer bricks.
	 * This should at least catch the most egregious errors. */
	for(size_t r=0; r < nreq; ++r) {
		for(size_t dim=0; dim < 4; ++dim) {
			if(requests[r*4+dim] >= bdims[dim]) {
				if(erridx != NULL) { *erridx = r; }
				return false;
			}
		}
	}
	return true;
}

int
main(int argc, char* argv[])
{
	argparse(argc, argv);

	const size_t N_ht = htN();
	const unsigned main_brickdims[4] = { bricksX(), bricksY(), bricksZ(),
	                                     LODs() };

	hipError_t cerr = hipMemcpyToSymbol(HIP_SYMBOL(brickdims), main_brickdims,
	                                      sizeof(unsigned)*4, 0,
	                                      hipMemcpyHostToDevice);
	if(cerr != hipSuccess) {
		fprintf(stderr, "could not copy brickdim data: %s\n",
		        hipGetErrorString(cerr));
		exit(EXIT_FAILURE);
	}

	fprintf(stderr, "%zu-element hash table.\n", N_ht);
	/* create our hash table, and a chunk of memory to read it back into
	 * when we're done.  We could also use pinned memory.. but...
	 * well, we should try that, too. */
	unsigned* htable_dev = NULL;
	hipError_t err = hipMalloc(&htable_dev,
	                             N_ht*sizeof(unsigned));
	if(err != hipSuccess) {
		fprintf(stderr, "dev alloc of HT (size %zu) failed!: %s.\n",
		        N_ht, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	fprintf(stderr, "dev alloc ht okay\n");

	unsigned* htable_host = (unsigned*)calloc(sizeof(unsigned), N_ht);
	assert(htable_host); /* sometimes I <3 not being a real SW developer. */

	/* copy our (empty) hash table to the device: initialized to all 0s. */
	err = hipMemcpy(htable_dev, htable_host,
	                 N_ht*sizeof(unsigned), hipMemcpyHostToDevice);
	if(err != hipSuccess) {
		fprintf(stderr, "cuda copy error host -> dev: %s\n",
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	size_t nrequests;
	unsigned* bricks_host = requests_from(requestfile(), &nrequests);
	if(bricks_host == NULL) {
		fprintf(stderr, "Could not read requests from %s!\n",
		        requestfile());
		exit(EXIT_FAILURE);
	}
	size_t fault;
	if(!requests_verify(bricks_host, nrequests, main_brickdims, &fault)) {
		fprintf(stderr, "Brick request %zu is garbage.\n", fault);
		exit(EXIT_FAILURE);
	}
	const size_t brickbytes = nrequests * 4 * sizeof(uint32_t);

	uint32_t* bricks_dev;
	/* each brick request is 16 bytes: 4 unsigned numbers (X,Y,Z,LOD) */
	err = hipMalloc(&bricks_dev, brickbytes);
	if(err != hipSuccess) {
		fprintf(stderr, "cuda alloc error for bricks (dev): %s\n",
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	fprintf(stderr, "dev alloc bricks okay\n");

	err = hipMemcpy(bricks_dev, bricks_host, brickbytes,
	                 hipMemcpyHostToDevice);
	if(err != hipSuccess) {
		fprintf(stderr, "cuda copy error (bricks) host -> dev: %s\n",
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	fprintf(stderr, "bricks cpy okay\n");

	dim3 blocks(60, 33);
#if 1
	ht_inserts<<<blocks, 32>>>(htable_dev, N_ht, bricks_dev, nrequests);
#else
	ht_inserts_simple<<<blocks, 32>>>(htable_dev, N_ht, bricks_dev,
	                                  nrequests);
#endif

	if((err = hipGetLastError()) != hipSuccess) {
		fprintf(stderr, "Failed to launch kernel: %s\n",
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	/* get the hash table back. */
	err = hipMemcpy(htable_host, htable_dev,
	                 N_ht*sizeof(unsigned),
	                 hipMemcpyDeviceToHost);
	if(err != hipSuccess) {
		fprintf(stderr, "copy error (htable) dev -> host: %s\n",
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	printf("Test PASSED\n");

	if((err = hipFree(htable_dev)) != hipSuccess) {
		fprintf(stderr, "couldn't free device hash table! %s\n",
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	free(htable_host);

	/* needed for benchmarks, though the reset afterwards probably makes it
	 * irrelevant. */
	hipDeviceSynchronize();
	if((err = hipDeviceReset()) != hipSuccess) {
		fprintf(stderr, "Failed to deinitialize the device! error=%s\n",
		        hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	return 0;
}
